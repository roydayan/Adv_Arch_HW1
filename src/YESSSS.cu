#include "hip/hip_runtime.h"

//0.061 ms
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_DIM 32
#define THREADS_PER_DIM 16
#define MAT_SIZE 1024
#define BATCH_SIZE 10000
#define MAT_PER_BLOCK 1
#define TOTAL_BLOCKS 10000 // (BATCH_SIZE / MAT_PER_BLOCK)

__global__ void matmul_kernel_optimized(const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ out) {

    int batch_idx = blockIdx.x;;
    int row = threadIdx.y * 2;
    int col = threadIdx.x * 2;    

    // Better memory coalescing (threads in a warp access consecutive memory):
    int entry_offset = batch_idx * MAT_SIZE + row * TILE_DIM + col;
    __shared__ float shared_a[TILE_DIM][TILE_DIM];
    __shared__ float shared_b[TILE_DIM][TILE_DIM];

    shared_a[row][col] = a[entry_offset];
    shared_a[row][col + 1] = a[entry_offset + 1];
    shared_a[row + 1][col] = a[entry_offset + TILE_DIM];
    shared_a[row + 1][col + 1] = a[entry_offset + TILE_DIM + 1];
    shared_b[row][col] = b[entry_offset];
    shared_b[row][col + 1] = b[entry_offset + 1];
    shared_b[row + 1][col] = b[entry_offset + TILE_DIM];
    shared_b[row + 1][col + 1] = b[entry_offset + TILE_DIM + 1];

    __syncthreads();

    /* for one entry:
    float sum = 0.0f;
    for (int k = 0; k < TILE_DIM; ++k) {
        sum += shared_a[row][k] * shared_b[k][col];
    }
    out[entry_offset] = sum;
    */
    // for 4 entries:
    float c_sum[4] = {0.f, 0.f, 0.f, 0.f};
    float a0;
    float a1;
    float b0;
    float b1;
    #pragma unroll 32
    for (int k = 0; k < TILE_DIM; ++k) {
        a0 = shared_a[row][k];
        a1 = shared_a[row + 1][k];
        b0 = shared_b[k][col];
        b1 = shared_b[k][col + 1];

        c_sum[0] += a0 * b0;
        c_sum[1] += a0 * b1;
        c_sum[2] += a1 * b0;
        c_sum[3] += a1 * b1;
    }

    // write back
    int out_entry = batch_idx * MAT_SIZE + row * TILE_DIM + col;
    out[out_entry] = c_sum[0];
    out[out_entry + 1] = c_sum[1];
    out[out_entry + TILE_DIM] = c_sum[2];
    out[out_entry + TILE_DIM + 1] = c_sum[3];
}

void matmul_cuda_forward(torch::Tensor a1, torch::Tensor a2, torch::Tensor out) {
    const dim3 threads(THREADS_PER_DIM, THREADS_PER_DIM);   // 16×16 = 256
    const dim3 blocks(TOTAL_BLOCKS);

    matmul_kernel_optimized<<<blocks, threads>>>(
        a1.data_ptr<float>(),
        a2.data_ptr<float>(),
        out.data_ptr<float>()
    );
    //for (int i=0; i<500000000;i++){}; // busy wait
}
